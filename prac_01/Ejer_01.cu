#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

int main() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    if (err != hipSuccess) {
        cerr << "Error al obtener el número de dispositivos: " << hipGetErrorString(err) << endl;
        return -1;
    }

    cout << "Número de dispositivos CUDA: " << deviceCount << endl;

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        cout << "\n=== Dispositivo " << i << " ===" << endl;
        cout << "Nombre: " << prop.name << endl;
        cout << "Capacidad de cómputo: " << prop.major << "." << prop.minor << endl;
        cout << "Memoria global total: " << (prop.totalGlobalMem / (1024 * 1024)) << " MB" << endl;
        cout << "Número de multiprocesadores: " << prop.multiProcessorCount << endl;
        cout << "Tamaño de warp: " << prop.warpSize << endl;
        cout << "Memoria compartida por bloque: " << prop.sharedMemPerBlock << " bytes" << endl;
        cout << "Hilos máximos por bloque: " << prop.maxThreadsPerBlock << endl;
        cout << "Tamaño máximo de bloques (x,y,z): ("
                  << prop.maxThreadsDim[0] << ", "
                  << prop.maxThreadsDim[1] << ", "
                  << prop.maxThreadsDim[2] << ")" << endl;
        cout << "Tamaño máximo de grid (x,y,z): ("
                  << prop.maxGridSize[0] << ", "
                  << prop.maxGridSize[1] << ", "
                  << prop.maxGridSize[2] << ")" << endl;
    }

    return 0;
}
