
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <vector>

#define N 10000
#define THREADS_PER_BLOCK 256

__global__ void vectorAdd(float *A, float *B, float *C, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    std::srand(static_cast<unsigned>(std::time(0)));

    // Vectores en host
    std::vector<float> h_A(N);
    std::vector<float> h_B(N);
    std::vector<float> h_C(N); // resultado

    // Inicializar con valores aleatorios
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Punteros en device
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    // Copiar datos al device
    hipMemcpy(d_A, h_A.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), N * sizeof(float), hipMemcpyHostToDevice);

    // Lanzar kernel
    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    vectorAdd<<<blocks, THREADS_PER_BLOCK>>>(d_A, d_B, d_C, N);

    // Copiar resultado al host
    hipMemcpy(h_C.data(), d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Mostrar algunos resultados
    std::cout << "A[0] + B[0] = " << h_A[0] << " + " << h_B[0] << " = " << h_C[0] << std::endl;

    // Liberar memoria en device
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
