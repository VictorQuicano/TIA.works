
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>

#define M 2  // filas de A y C
#define K 2  // columnas de A y filas de B
#define N 2  // columnas de B y C
#define THREADS_PER_BLOCK 16

__global__ void matrixMulKernel(int* A, int* B, int* C, int m, int k, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += A[row * k + i] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    std::srand(static_cast<unsigned>(std::time(0)));

    // Host matrices
    std::vector<int> h_A(M * K);
    std::vector<int> h_B(K * N);
    std::vector<int> h_C(M * N);

    // Inicializar matrices con valores aleatorios
    for (int i = 0; i < M * K; ++i) h_A[i] = static_cast<int>(rand()) % 21;
    for (int i = 0; i < K * N; ++i) h_B[i] = static_cast<int>(rand()) % 21;

    // Device matrices
    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    // Copiar datos al device
    hipMemcpy(d_A, h_A.data(), M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), K * N * sizeof(float), hipMemcpyHostToDevice);

    // Definir bloques e hilos
    dim3 blockDim(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 gridDim((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
                 (M + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    // Ejecutar kernel
    matrixMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);

    // Copiar resultado al host
    hipMemcpy(h_C.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    if(M < 10 && N < 10 && K < 10) {
        // Mostrar matrices
        std::cout << "Matriz A:" << std::endl;
        for (int i = 0; i < M; ++i) {
            for (int j = 0; j < K; ++j) {
                std::cout << h_A[i * K + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << "Matriz B:" << std::endl;
        for (int i = 0; i < K; ++i) {
            for (int j = 0; j < N; ++j) {
                std::cout << h_B[i * N + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << "Resultado de la multiplicación de matrices:" << std::endl;
        for (int i = 0; i < M; ++i) {
            for (int j = 0; j < N; ++j) {
                std::cout << h_C[i * N + j] << " ";
            }
            std::cout << std::endl;
        }
    } else {
        std::cout << "Resultado de la multiplicación de matrices:" << std::endl;
        for (int i = 0; i < M * N; ++i) {
            std::cout << h_C[i] << " ";
        }
        std::cout << std::endl;   
    }

    // Liberar memoria del device
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
